#include "hip/hip_runtime.h"
/** @file   testbed_geometry.cu
 *  @author Fatemeh Salehi
 */

#include <neural-graphics-primitives/common_device.cuh>
#include <neural-graphics-primitives/common.h>
#include <neural-graphics-primitives/discrete_distribution.h>
#include <neural-graphics-primitives/envmap.cuh>
#include <neural-graphics-primitives/random_val.cuh> // helpers to generate random values, directions
#include <neural-graphics-primitives/render_buffer.h>
#include <neural-graphics-primitives/takikawa_encoding.cuh>
#include <neural-graphics-primitives/testbed.h>
#include <neural-graphics-primitives/geometry.h>
#include <neural-graphics-primitives/tinyobj_loader_wrapper.h>
#include <neural-graphics-primitives/trainable_buffer.cuh>
#include <neural-graphics-primitives/triangle_bvh.cuh>
#include <neural-graphics-primitives/triangle_octree.cuh>
#include <neural-graphics-primitives/geometry_bvh.cuh>

#include <tiny-cuda-nn/encodings/grid.h>
#include <tiny-cuda-nn/gpu_matrix.h>
#include <tiny-cuda-nn/network_with_input_encoding.h>
#include <tiny-cuda-nn/trainer.h>

namespace ngp {


// TODO: all m_aabb s should be changed to local/node bounding boxes and passed/stored in the node 



// struct MeshPayload {
// 	vec3 dir; // direction of the ray
// 	uint32_t idx; // index of the ray
// 	uint16_t n_steps;
// 	bool alive;
// 	// uint32_t triangle_id; // ID of the triangle that the ray hit
//     // vec3 hit_point; // point where the ray hit the triangle
//     // vec3 normal; // normal of the triangle at the hit point

// };

// struct RaysMeshSoa {	// most probably this is not correct
// #if defined(__HIPCC__) || (defined(__clang__) && defined(__CUDA__))
//     void copy_from_other_async(uint32_t n_elements, const RaysMeshSoa& other, hipStream_t stream) {
//         CUDA_CHECK_THROW(hipMemcpyAsync(origin, other.origin, n_elements * sizeof(vec3), hipMemcpyDeviceToDevice, stream));
//         CUDA_CHECK_THROW(hipMemcpyAsync(dir, other.dir, n_elements * sizeof(vec3), hipMemcpyDeviceToDevice, stream));
//         CUDA_CHECK_THROW(hipMemcpyAsync(payload, other.payload, n_elements * sizeof(MeshPayload), hipMemcpyDeviceToDevice, stream));
//     }
// #endif

//     void set(vec3* origin, vec3* dir, MeshPayload* payload) {
//         this->origin = origin;
//         this->dir = dir;
//         this->payload = payload;
//     }

//     vec3* origin;	//maybe pos?
//     vec3* dir;
//     MeshPayload* payload;
// };


__global__ void shade_kernel_geometry(
	const uint32_t n_elements,
	BoundingBox aabb,
	float floor_y,
	const ERenderMode mode::Shade,
	const BRDFParams brdf,
	vec3 sun_dir,
	vec3 up_dir,
	mat4x3 camera_matrix,
	vec3* __restrict__ positions,
	vec3* __restrict__ normals,
	float* __restrict__ distances,
	geometryPayload* __restrict__ payloads,
	vec4* __restrict__ frame_buffer,
	float* __restrict__ depth_buffer
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	GeometryPayload& payload = payloads[i];
	if (!aabb.contains(positions[i])) {
		return;
	}

	// The normal in memory isn't normalized yet
	vec3 normal = normalize(normals[i]);
	vec3 pos = positions[i];
	bool floor = false;
	if (pos.y < floor_y + 0.001f && payload.dir.y < 0.f) {
		normal = vec3{0.0f, 1.0f, 0.0f};
		floor = true;
	}

	vec3 cam_pos = camera_matrix[3];
	vec3 cam_fwd = camera_matrix[2];
	float ao = powf(0.92f, payload.n_steps * 0.5f) * (1.f / 0.92f);
	vec3 color;

	float skyam = -dot(normal, up_dir) * 0.5f + 0.5f;
	vec3 suncol = vec3{255.f/255.0f, 225.f/255.0f, 195.f/255.0f} * 4.f * distances[i]; // Distance encodes shadow occlusion. 0=occluded, 1=no shadow
	const vec3 skycol = vec3{195.f/255.0f, 215.f/255.0f, 255.f/255.0f} * 4.f * skyam;
	float check_size = 8.f/aabb.diag().x;
	float check=((int(floorf(check_size*(pos.x-aabb.min.x)))^int(floorf(check_size*(pos.z-aabb.min.z)))) &1) ? 0.8f : 0.2f;
	const vec3 floorcol = vec3{check*check*check, check*check, check};
	color = evaluate_shading(
		floor ? floorcol : brdf.basecolor * brdf.basecolor,
		brdf.ambientcolor * skycol,
		suncol,
		floor ? 0.f : brdf.metallic,
		floor ? 0.f : brdf.subsurface,
		floor ? 1.f : brdf.specular,
		floor ? 0.5f : brdf.roughness,
		0.f,
		floor ? 0.f : brdf.sheen,
		0.f,
		floor ? 0.f : brdf.clearcoat,
		brdf.clearcoat_gloss,
		sun_dir,
		-normalize(payload.dir),
		normal
	);

	frame_buffer[payload.idx] = {color.r, color.g, color.b, 1.0f};
	depth_buffer[payload.idx] = dot(cam_fwd, pos - cam_pos);
}



// separates the "alive" and "dead" elements of the input arrays into two separate arrays
__global__ void compact_kernel_geometry(
	const uint32_t n_elements,
	const float zero_offset,
	vec3* src_positions, float* src_distances, GeometryPayload* src_payloads,
	vec3* dst_positions, float* dst_distances, GeometryPayload* dst_payloads,
	vec3* dst_final_positions, float* dst_final_distances, GeometryPayload* dst_final_payloads,
	BoundingBox aabb,
	uint32_t* counter, uint32_t* finalCounter
) {
	const uint32_t i = threadIdx.x + blockIdx.x * blockDim.x;
	if (i >= n_elements) return;

	GeometryPayload& src_payload = src_payloads[i];

	if (src_payload.alive) {
		uint32_t idx = atomicAdd(counter, 1);
		dst_payloads[idx] = src_payload;
		dst_positions[idx] = src_positions[i];
		dst_distances[idx] = src_distances[i];
	} else if (aabb.contains(src_positions[i])) {
		uint32_t idx = atomicAdd(finalCounter, 1);
		dst_final_payloads[idx] = src_payload;
		dst_final_positions[idx] = src_positions[i];
		dst_final_distances[idx] = 1.0f; // HACK: Distances encode shadowing factor when shading
	}
}

__global__ void init_rays_with_payload_kernel_geometry(
	uint32_t sample_index,
	vec3* __restrict__ positions,
	float* __restrict__ distances,
	GeometryPayload* __restrict__ payloads,
	ivec2 resolution,
	vec2 focal_length,
	mat4x3 camera_matrix,
	vec2 screen_center,
	vec3 parallax_shift,
	bool snap_to_pixel_centers,
	BoundingBox aabb,
	float floor_y,
	float near_distance,
	float plane_z,
	float aperture_size,
	Foveation foveation,
	Buffer2DView<const vec4> envmap,
	vec4* __restrict__ frame_buffer,
	float* __restrict__ depth_buffer,
	Buffer2DView<const uint8_t> hidden_area_mask,
	const TriangleOctreeNode* __restrict__ octree_nodes = nullptr,
	int max_octree_depth = 0
) {
	uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x || y >= resolution.y) {
		return;
	}

	uint32_t idx = x + resolution.x * y;

	if (plane_z < 0) {
		aperture_size = 0.0;
	}

	Ray ray = pixel_to_ray(
		sample_index,
		{(int)x, (int)y},
		resolution,
		focal_length,
		camera_matrix,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		near_distance,
		plane_z,
		aperture_size,
		foveation,
		hidden_area_mask
	);

	distances[idx] = MAX_DEPTH();
	depth_buffer[idx] = MAX_DEPTH();

	GeometryPayload& payload = payloads[idx];

	if (!ray.is_valid()) {
		payload.dir = ray.d;
		payload.idx = idx;
		payload.n_steps = 0;
		payload.alive = false;
		positions[idx] = ray.o;
		return;
	}

	if (plane_z < 0) {
		float n = length(ray.d);
		payload.dir = (1.0f/n) * ray.d;
		payload.idx = idx;
		payload.n_steps = 0;
		payload.alive = false;
		positions[idx] = ray.o - plane_z * ray.d;
		depth_buffer[idx] = -plane_z;
		return;
	}

	ray.d = normalize(ray.d);
	float t = max(aabb.ray_intersect(ray.o, ray.d).x, 0.0f);

	ray.advance(t + 1e-6f);

	if (octree_nodes && !TriangleOctree::contains(octree_nodes, max_octree_depth, ray.o)) {
		t = max(0.0f, TriangleOctree::ray_intersect(octree_nodes, max_octree_depth, ray.o, ray.d));
		if (ray.o.y > floor_y && ray.d.y < 0.f) {
			float floor_dist = -(ray.o.y - floor_y) / ray.d.y;
			if (floor_dist > 0.f) {
				t = min(t, floor_dist);
			}
		}

		ray.advance(t + 1e-6f);
	}

	positions[idx] = ray.o;

	// if (envmap) {
	// 	frame_buffer[idx] = read_envmap(envmap, ray.d);
	// }

	payload.dir = ray.d;
	payload.idx = idx;
	payload.n_steps = 0;
	payload.alive = aabb.contains(ray.o);
}


__global__ void init_rays_with_payload_kernel_geometry(
    uint32_t sample_index,
    vec3* __restrict__ positions,
    float* __restrict__ distances,
    GeometryPayload* __restrict__ payloads,
    ivec2 resolution,
    vec2 focal_length,
    mat4x3 camera_matrix,
    vec2 screen_center,
    vec3 parallax_shift,
    bool snap_to_pixel_centers,
    BoundingBox aabb,
    float floor_y,
    float near_distance,
    float plane_z,
    float aperture_size,
    Foveation foveation,
    Buffer2DView<const vec4> envmap,
    vec4* __restrict__ frame_buffer,
    float* __restrict__ depth_buffer,
    Buffer2DView<const uint8_t> hidden_area_mask,
    const TriangleOctreeNode* __restrict__ octree_nodes = nullptr,
    int max_octree_depth = 0
) {
    uint32_t x = threadIdx.x + blockDim.x * blockIdx.x;
	uint32_t y = threadIdx.y + blockDim.y * blockIdx.y;

	if (x >= resolution.x || y >= resolution.y) {
		return;
	}

	uint32_t idx = x + resolution.x * y;

	if (plane_z < 0) {
		aperture_size = 0.0;
	}

	Ray ray = pixel_to_ray(
		sample_index,
		{(int)x, (int)y},
		resolution,
		focal_length,
		camera_matrix,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		near_distance,
		plane_z,
		aperture_size,
		foveation,
		hidden_area_mask
	);

	distances[idx] = MAX_DEPTH();	// I don't know what this is for	
	depth_buffer[idx] = MAX_DEPTH();

    GeometryPayload& payload = payloads[idx];

    if (!ray.is_valid()) {
		payload.dir = ray.d;
		payload.idx = idx;
		payload.n_steps = 0;
		payload.alive = false;
		positions[idx] = ray.o;
		return;
	}

	if (plane_z < 0) {
		float n = length(ray.d);
		payload.dir = (1.0f/n) * ray.d;
		payload.idx = idx;
		payload.n_steps = 0;
		payload.alive = false;
		positions[idx] = ray.o - plane_z * ray.d;
		depth_buffer[idx] = -plane_z;
		return;
	}

	ray.d = normalize(ray.d);
    float t = max(aabb.ray_intersect(ray.o, ray.d).x, 0.0f);

	ray.advance(t + 1e-6f);

    if (octree_nodes && !TriangleOctree::contains(octree_nodes, max_octree_depth, ray.o)) {
        t = max(0.0f, TriangleOctree::ray_intersect(octree_nodes, max_octree_depth, ray.o, ray.d));
        if (ray.o.y > floor_y && ray.d.y < 0.f) {
			float floor_dist = -(ray.o.y - floor_y) / ray.d.y;
			if (floor_dist > 0.f) {
				t = min(t, floor_dist);
			}
		}

		ray.advance(t + 1e-6f);
    }

    positions[idx] = ray.o;

	if (envmap) {
		frame_buffer[idx] = read_envmap(envmap, ray.d);
	}

    payload.dir = ray.d;
    payload.idx = idx;
    payload.n_steps = 0;
    payload.alive = aabb.contains(ray.o);
}

void Testbed::MyTracer::init_rays_from_camera(
	uint32_t sample_index,
	const ivec2& resolution,
	const vec2& focal_length,
	const mat4x3& camera_matrix,
	const vec2& screen_center,
	const vec3& parallax_shift,
	bool snap_to_pixel_centers,
	const BoundingBox& aabb,
	float floor_y,
	float near_distance,
	float plane_z,
	float aperture_size,
	const Foveation& foveation,
	const Buffer2DView<const vec4>& envmap,
	vec4* frame_buffer,
	float* depth_buffer,
	const Buffer2DView<const uint8_t>& hidden_area_mask,
	const TriangleOctree* octree,
	uint32_t n_octree_levels,
	hipStream_t stream
) {
	// Make sure we have enough memory reserved to render at the requested resolution
	size_t n_pixels = (size_t)resolution.x * resolution.y;
	enlarge(n_pixels, stream);

	// defining the grid and block dimensions for launching the CUDA kernel
	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)resolution.x, threads.x), div_round_up((uint32_t)resolution.y, threads.y), 1 };
	init_rays_with_payload_kernel_geometry<<<blocks, threads, 0, stream>>>(
		sample_index,
		m_rays[0].pos,
		m_rays[0].distance,
		m_rays[0].payload,
		resolution,
		focal_length,
		camera_matrix,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		aabb,
		floor_y,
		near_distance,
		plane_z,
		aperture_size,
		foveation,
		envmap,
		frame_buffer,
		depth_buffer,
		hidden_area_mask,
		octree ? octree->nodes_gpu() : nullptr,
		octree ? n_octree_levels : 0
	);
	m_n_rays_initialized = (uint32_t)n_pixels;
}

uint32_t Testbed::MyTracer::trace_bvh(TriangleBvh* bvh, const Triangle* triangles, hipStream_t stream) {
	uint32_t n_alive = m_n_rays_initialized;
	m_n_rays_initialized = 0;

	if (!bvh) {
		return 0;
	}

	// Abuse the normal buffer to temporarily hold ray directions
	parallel_for_gpu(stream, n_alive, [payloads=m_rays[0].payload, normals=m_rays[0].normal] __device__ (size_t i) {
		normals[i] = payloads[i].dir;
	});

	//  if optix is available, uses optix.raytrace->invoke
	bvh->ray_trace_gpu(n_alive, m_rays[0].pos, m_rays[0].normal, triangles, stream);
	return n_alive;
}

uint32_t Testbed::MyTracer::trace(
    const Triangle* triangles,
    uint32_t num_triangles,
    float zero_offset,
    float distance_scale,
    float maximum_distance,
    const BoundingBox& aabb,
    const float floor_y,
    const TriangleOctree* octree,
    const uint32_t n_octree_levels,
    hipStream_t stream
) {
    if (m_n_rays_initialized == 0) {
        return 0;
    }

    CUDA_CHECK_THROW(hipMemsetAsync(m_hit_counter, 0, sizeof(uint32_t), stream));

    const uint32_t STEPS_INBETWEEN_COMPACTION = 4;

    uint32_t n_alive = m_n_rays_initialized;
    m_n_rays_initialized = 0;

    uint32_t i = 1;
    uint32_t double_buffer_index = 0;
    while (i < MARCH_ITER) {
        uint32_t step_size = std::min(i, STEPS_INBETWEEN_COMPACTION);

		// double buffer
        GeometryPayload& rays_current = m_rays[(double_buffer_index+1)%2];
        GeometryPayload& rays_tmp = m_rays[double_buffer_index%2];
        ++double_buffer_index;

        // Compact rays that did not diverge yet
        {
            CUDA_CHECK_THROW(hipMemsetAsync(m_alive_counter, 0, sizeof(uint32_t), stream));
            linear_kernel(compact_kernel_geometry, 0, stream,
                n_alive,
                zero_offset,
                rays_tmp.pos, rays_tmp.distance, rays_tmp.payload,
                rays_current.pos, rays_current.distance, rays_current.payload,
                m_rays_hit.pos, m_rays_hit.distance, m_rays_hit.payload,
                aabb,
                m_alive_counter, m_hit_counter
            );
            CUDA_CHECK_THROW(hipMemcpyAsync(&n_alive, m_alive_counter, sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
            CUDA_CHECK_THROW(hipStreamSynchronize(stream));
        }

        if (n_alive == 0) {
            break;
        }

        for (uint32_t j = 0; j < step_size; ++j) {
            linear_kernel(ray_triangle_intersection_kernel, 0, stream,
                n_alive,
                rays_current.pos,
                rays_current.distance,
                rays_current.payload,
                aabb,
                floor_y,
                triangles,
                num_triangles,
                distance_scale,
                maximum_distance
            );
        }

        i += step_size;
    }

    uint32_t n_hit;
    CUDA_CHECK_THROW(hipMemcpyAsync(&n_hit, m_hit_counter, sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
    CUDA_CHECK_THROW(hipStreamSynchronize(stream));
    return n_hit;
}

// allocate and distribute workspace memory for rays
void Testbed::MyTracer::enlarge(size_t n_elements, hipStream_t stream) {
	n_elements = next_multiple(n_elements, size_t(BATCH_SIZE_GRANULARITY));
	auto scratch = allocate_workspace_and_distribute<
		vec3, vec3, float, float, float, float, GeometryPayload, // m_rays[0]
		vec3, vec3, float, float, float, float, GeometryPayload, // m_rays[1]
		vec3, vec3, float, float, float, float, GeometryPayload, // m_rays_hit

		uint32_t,
		uint32_t
	>(
		stream, &m_scratch_alloc,
		n_elements, n_elements, n_elements, n_elements, n_elements, n_elements, n_elements,
		n_elements, n_elements, n_elements, n_elements, n_elements, n_elements, n_elements,
		n_elements, n_elements, n_elements, n_elements, n_elements, n_elements, n_elements,
		32, // 2 full cache lines to ensure no overlap
		32  // 2 full cache lines to ensure no overlap
	);

	m_rays[0].set(std::get<0>(scratch), std::get<1>(scratch), std::get<2>(scratch), std::get<3>(scratch), std::get<4>(scratch), std::get<5>(scratch), std::get<6>(scratch));
	m_rays[1].set(std::get<7>(scratch), std::get<8>(scratch), std::get<9>(scratch), std::get<10>(scratch), std::get<11>(scratch), std::get<12>(scratch), std::get<13>(scratch));
	m_rays_hit.set(std::get<14>(scratch), std::get<15>(scratch), std::get<16>(scratch), std::get<17>(scratch), std::get<18>(scratch), std::get<19>(scratch), std::get<20>(scratch));

	m_hit_counter = std::get<21>(scratch);
	m_alive_counter = std::get<22>(scratch);
}



// todo: tracer for nerfs and tracer for the mesh based objects
// init_rays_from_camera, init_rays_from_data, enlarge
void Testbed::render_geometry(
	hipStream_t stream,
		CudaDevice& device,
		const CudaRenderBufferView& render_buffer,
		const vec2& focal_length,
		const mat4x3& camera_matrix,
		const vec2& screen_center,
		const Foveation& foveation,
		int visualized_dimension
) {
	float plane_z = m_slice_plane_z + m_scale;
	float distance_scale = 1.f/std::max(m_volume.inv_distance_scale,0.01f);
	auto res = render_buffer.resolution;
}

void Testbed::render_mesh(
    hipStream_t stream,
	const distance_fun_t& distance_function,
	const normals_fun_t& normals_function,
	const CudaRenderBufferView& render_buffer,
	const vec2& focal_length,
	const mat4x3& camera_matrix,
	const vec2& screen_center,
	const Foveation& foveation,
	int visualized_dimension,
	const BoundingBox& aabb
) {
	float plane_z = m_slice_plane_z + m_scale;
	
	auto* octree_ptr = m_meshData.triangle_octree.get();

	MyTracer tracer;

	uint32_t n_octree_levels = octree_ptr ? octree_ptr->depth() : 0;

	BoundingBox mesh_bounding_box = aabb;
	mesh_bounding_box.inflate(m_meshData.zero_offset);
	tracer.init_rays_from_camera(
		render_buffer.spp,
		render_buffer.resolution,
		focal_length,
		camera_matrix,
		screen_center,
		m_parallax_shift,
		m_snap_to_pixel_centers,
		mesh_bounding_box,
		get_floor_y(),
		m_render_near_distance,
		plane_z,
		m_aperture_size,
		foveation,
		m_envmap.inference_view(),
		render_buffer.frame_buffer,
		render_buffer.depth_buffer,
		render_buffer.hidden_area_mask ? render_buffer.hidden_area_mask->const_view() : Buffer2DView<const uint8_t>{},
		octree_ptr,
		n_octree_levels,
		stream
	);

	auto trace = [&](MyTracer& tracer) {
		return tracer.trace_bvh(m_meshData.triangle_bvh.get(), m_meshData.triangles_gpu.data(), stream);
	};

	uint32_t n_hit = trace(tracer);

	RaysMeshSoa& rays_hit = tracer.rays_hit();

	normals_function(n_hit, rays_hit.pos, rays_hit.normal, stream);

	linear_kernel(shade_kernel_geometry, 0, stream,
		n_hit,
		aabb,
		get_floor_y(),
		render_mode,
		m_meshData.brdf,
		normalize(m_sun_dir),
		normalize(m_up_dir),
		camera_matrix,
		rays_hit.pos,
		rays_hit.normal,
		rays_hit.distance,
		rays_hit.payload,
		render_buffer.frame_buffer,
		render_buffer.depth_buffer
	);
}
// 

// /****************************************************/
// /*  The following code is from testbed_nerf.cu file */

void Testbed::NerfTracer::init_rays_from_camera(
	uint32_t sample_index,
	uint32_t padded_output_width,
	uint32_t n_extra_dims,
	const ivec2& resolution,
	const vec2& focal_length,
	const mat4x3& camera_matrix0,
	const mat4x3& camera_matrix1,
	const vec4& rolling_shutter,
	const vec2& screen_center,
	const vec3& parallax_shift,
	bool snap_to_pixel_centers,
	const BoundingBox& render_aabb,
	const mat3& render_aabb_to_local,
	float near_distance,
	float plane_z,
	float aperture_size,
	const Foveation& foveation,
	const Lens& lens,
	const Buffer2DView<const vec4>& envmap,
	const Buffer2DView<const vec2>& distortion,
	vec4* frame_buffer,
	float* depth_buffer,
	const Buffer2DView<const uint8_t>& hidden_area_mask,
	const uint8_t* grid,
	int show_accel,
	uint32_t max_mip,
	float cone_angle_constant,
	ERenderMode render_mode,
	hipStream_t stream
) {
	// Make sure we have enough memory reserved to render at the requested resolution
	size_t n_pixels = (size_t)resolution.x * resolution.y;
	enlarge(n_pixels, padded_output_width, n_extra_dims, stream);

	const dim3 threads = { 16, 8, 1 };
	const dim3 blocks = { div_round_up((uint32_t)resolution.x, threads.x), div_round_up((uint32_t)resolution.y, threads.y), 1 };
	init_rays_with_payload_kernel_nerf<<<blocks, threads, 0, stream>>>(
		sample_index,
		m_rays[0].payload,
		resolution,
		focal_length,
		camera_matrix0,
		camera_matrix1,
		rolling_shutter,
		screen_center,
		parallax_shift,
		snap_to_pixel_centers,
		render_aabb,
		render_aabb_to_local,
		near_distance,
		plane_z,
		aperture_size,
		foveation,
		lens,
		envmap,
		frame_buffer,
		depth_buffer,
		hidden_area_mask,
		distortion,
		render_mode
	);

	m_n_rays_initialized = resolution.x * resolution.y;

	CUDA_CHECK_THROW(hipMemsetAsync(m_rays[0].rgba, 0, m_n_rays_initialized * sizeof(vec4), stream));
	CUDA_CHECK_THROW(hipMemsetAsync(m_rays[0].depth, 0, m_n_rays_initialized * sizeof(float), stream));

	linear_kernel(advance_pos_nerf_kernel, 0, stream,
		m_n_rays_initialized,
		render_aabb,
		render_aabb_to_local,
		camera_matrix1[2],
		focal_length,
		sample_index,
		m_rays[0].payload,
		grid,
		(show_accel >= 0) ? show_accel : 0,
		max_mip,
		cone_angle_constant
	);
}

uint32_t Testbed::NerfTracer::trace(
	const std::shared_ptr<NerfNetwork<network_precision_t>>& network,
	const BoundingBox& render_aabb,
	const mat3& render_aabb_to_local,
	const BoundingBox& train_aabb,
	const vec2& focal_length,
	float cone_angle_constant,
	const uint8_t* grid,
	ERenderMode render_mode,
	const mat4x3 &camera_matrix,
	float depth_scale,
	int visualized_layer,
	int visualized_dim,
	ENerfActivation rgb_activation,
	ENerfActivation density_activation,
	int show_accel,
	uint32_t max_mip,
	float min_transmittance,
	float glow_y_cutoff,
	int glow_mode,
	const float* extra_dims_gpu,
	hipStream_t stream
) {
	if (m_n_rays_initialized == 0) {
		return 0;
	}

	CUDA_CHECK_THROW(hipMemsetAsync(m_hit_counter, 0, sizeof(uint32_t), stream));

	uint32_t n_alive = m_n_rays_initialized;
	// m_n_rays_initialized = 0;

	uint32_t i = 1;
	uint32_t double_buffer_index = 0;
	while (i < MARCH_ITER) {
		RaysNerfSoa& rays_current = m_rays[(double_buffer_index + 1) % 2];
		RaysNerfSoa& rays_tmp = m_rays[double_buffer_index % 2];
		++double_buffer_index;

		// Compact rays that did not diverge yet
		{
			CUDA_CHECK_THROW(hipMemsetAsync(m_alive_counter, 0, sizeof(uint32_t), stream));
			linear_kernel(compact_kernel_nerf, 0, stream,
				n_alive,
				rays_tmp.rgba, rays_tmp.depth, rays_tmp.payload,
				rays_current.rgba, rays_current.depth, rays_current.payload,
				m_rays_hit.rgba, m_rays_hit.depth, m_rays_hit.payload,
				m_alive_counter, m_hit_counter
			);
			CUDA_CHECK_THROW(hipMemcpyAsync(&n_alive, m_alive_counter, sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
			CUDA_CHECK_THROW(hipStreamSynchronize(stream));
		}

		if (n_alive == 0) {
			break;
		}

		// Want a large number of queries to saturate the GPU and to ensure compaction doesn't happen toooo frequently.
		uint32_t target_n_queries = 2 * 1024 * 1024;
		uint32_t n_steps_between_compaction = clamp(target_n_queries / n_alive, (uint32_t)MIN_STEPS_INBETWEEN_COMPACTION, (uint32_t)MAX_STEPS_INBETWEEN_COMPACTION);

		uint32_t extra_stride = network->n_extra_dims() * sizeof(float);
		PitchedPtr<NerfCoordinate> input_data((NerfCoordinate*)m_network_input, 1, 0, extra_stride);
		linear_kernel(generate_next_nerf_network_inputs, 0, stream,
			n_alive,
			render_aabb,
			render_aabb_to_local,
			train_aabb,
			focal_length,
			camera_matrix[2],
			rays_current.payload,
			input_data,
			n_steps_between_compaction,
			grid,
			(show_accel>=0) ? show_accel : 0,
			max_mip,
			cone_angle_constant,
			extra_dims_gpu
		);
		uint32_t n_elements = next_multiple(n_alive * n_steps_between_compaction, BATCH_SIZE_GRANULARITY);
		GPUMatrix<float> positions_matrix((float*)m_network_input, (sizeof(NerfCoordinate) + extra_stride) / sizeof(float), n_elements);
		GPUMatrix<network_precision_t, RM> rgbsigma_matrix((network_precision_t*)m_network_output, network->padded_output_width(), n_elements);
		network->inference_mixed_precision(stream, positions_matrix, rgbsigma_matrix);

		if (render_mode == ERenderMode::Normals) {
			network->input_gradient(stream, 3, positions_matrix, positions_matrix);
		} else if (render_mode == ERenderMode::EncodingVis) {
			network->visualize_activation(stream, visualized_layer, visualized_dim, positions_matrix, positions_matrix);
		}

		linear_kernel(composite_kernel_nerf, 0, stream,
			n_alive,
			n_elements,
			i,
			train_aabb,
			glow_y_cutoff,
			glow_mode,
			camera_matrix,
			focal_length,
			depth_scale,
			rays_current.rgba,
			rays_current.depth,
			rays_current.payload,
			input_data,
			m_network_output,
			network->padded_output_width(),
			n_steps_between_compaction,
			render_mode,
			grid,
			rgb_activation,
			density_activation,
			show_accel,
			min_transmittance
		);

		i += n_steps_between_compaction;
	}

	uint32_t n_hit;
	CUDA_CHECK_THROW(hipMemcpyAsync(&n_hit, m_hit_counter, sizeof(uint32_t), hipMemcpyDeviceToHost, stream));
	CUDA_CHECK_THROW(hipStreamSynchronize(stream));
	return n_hit;
}

void Testbed::NerfTracer::enlarge(size_t n_elements, uint32_t padded_output_width, uint32_t n_extra_dims, hipStream_t stream) {
	n_elements = next_multiple(n_elements, size_t(BATCH_SIZE_GRANULARITY));
	size_t num_floats = sizeof(NerfCoordinate) / sizeof(float) + n_extra_dims;
	auto scratch = allocate_workspace_and_distribute<
		vec4, float, NerfPayload, // m_rays[0]
		vec4, float, NerfPayload, // m_rays[1]
		vec4, float, NerfPayload, // m_rays_hit

		network_precision_t,
		float,
		uint32_t,
		uint32_t
	>(
		stream, &m_scratch_alloc,
		n_elements, n_elements, n_elements,
		n_elements, n_elements, n_elements,
		n_elements, n_elements, n_elements,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * padded_output_width,
		n_elements * MAX_STEPS_INBETWEEN_COMPACTION * num_floats,
		32, // 2 full cache lines to ensure no overlap
		32  // 2 full cache lines to ensure no overlap
	);

	m_rays[0].set(std::get<0>(scratch), std::get<1>(scratch), std::get<2>(scratch), n_elements);
	m_rays[1].set(std::get<3>(scratch), std::get<4>(scratch), std::get<5>(scratch), n_elements);
	m_rays_hit.set(std::get<6>(scratch), std::get<7>(scratch), std::get<8>(scratch), n_elements);

	m_network_output = std::get<9>(scratch);
	m_network_input = std::get<10>(scratch);

	m_hit_counter = std::get<11>(scratch);
	m_alive_counter = std::get<12>(scratch);
}

std::vector<float> Testbed::Nerf::Training::get_extra_dims_cpu(int trainview) const {
	if (dataset.n_extra_dims() == 0) {
		return {};
	}

	if (trainview < 0 || trainview >= dataset.n_images) {
		throw std::runtime_error{"Invalid training view."};
	}

	const float* extra_dims_src = extra_dims_gpu.data() + trainview * dataset.n_extra_dims();

	std::vector<float> extra_dims_cpu(dataset.n_extra_dims());
	CUDA_CHECK_THROW(hipMemcpy(extra_dims_cpu.data(), extra_dims_src, dataset.n_extra_dims() * sizeof(float), hipMemcpyDeviceToHost));

	return extra_dims_cpu;
}

void Testbed::Nerf::Training::update_extra_dims() {
	uint32_t n_extra_dims = dataset.n_extra_dims();
	std::vector<float> extra_dims_cpu(extra_dims_gpu.size());
	for (uint32_t i = 0; i < extra_dims_opt.size(); ++i) {
		const std::vector<float>& value = extra_dims_opt[i].variable();
		for (uint32_t j = 0; j < n_extra_dims; ++j) {
			extra_dims_cpu[i * n_extra_dims + j] = value[j];
		}
	}

	CUDA_CHECK_THROW(hipMemcpyAsync(extra_dims_gpu.data(), extra_dims_cpu.data(), extra_dims_opt.size() * n_extra_dims * sizeof(float), hipMemcpyHostToDevice));
}

void Testbed::render_nerf(
	hipStream_t stream,
	CudaDevice& device,
	const CudaRenderBufferView& render_buffer,
	const std::shared_ptr<NerfNetwork<network_precision_t>>& nerf_network,
	const uint8_t* density_grid_bitfield,
	const vec2& focal_length,
	const mat4x3& camera_matrix0,
	const mat4x3& camera_matrix1,
	const vec4& rolling_shutter,
	const vec2& screen_center,
	const Foveation& foveation,
	int visualized_dimension,
	const Nerf& nerf,
	const BoundingBox& aabb
) {
	float plane_z = m_slice_plane_z + m_scale;
	if (m_render_mode == ERenderMode::Slice) {
		plane_z = -plane_z;
	}

	ERenderMode render_mode = visualized_dimension > -1 ? ERenderMode::EncodingVis : m_render_mode;

	const float* extra_dims_gpu = nerf.get_rendering_extra_dims(stream);

	NerfTracer tracer;

	// Our motion vector code can't undo grid distortions -- so don't render grid distortion if DLSS is enabled.
	// (Unless we're in distortion visualization mode, in which case the distortion grid is fine to visualize.)
	auto grid_distortion =
		nerf.render_with_lens_distortion && (!m_dlss || m_render_mode == ERenderMode::Distortion) ?
		m_distortion.inference_view() :
		Buffer2DView<const vec2>{};

	Lens lens = nerf.render_with_lens_distortion ? nerf.render_lens : Lens{};

	auto resolution = render_buffer.resolution;

	tracer.init_rays_from_camera(
		render_buffer.spp,
		nerf_network->padded_output_width(),
		nerf_network->n_extra_dims(),
		render_buffer.resolution,
		focal_length,
		camera_matrix0,
		camera_matrix1,
		rolling_shutter,
		screen_center,
		m_parallax_shift,
		m_snap_to_pixel_centers,
		m_render_aabb,
		m_render_aabb_to_local,
		m_render_near_distance,
		plane_z,
		m_aperture_size,
		foveation,
		lens,
		m_envmap.inference_view(),
		grid_distortion,
		render_buffer.frame_buffer,
		render_buffer.depth_buffer,
		render_buffer.hidden_area_mask ? render_buffer.hidden_area_mask->const_view() : Buffer2DView<const uint8_t>{},
		density_grid_bitfield,
		nerf.show_accel,
		nerf.max_cascade,
		nerf.cone_angle_constant,
		render_mode,
		stream
	);

	float depth_scale = 1.0f / nerf.training.dataset.scale;
	bool render_2d = m_render_mode == ERenderMode::Slice || m_render_mode == ERenderMode::Distortion;

	uint32_t n_hit;
	if (render_2d) {
		n_hit = tracer.n_rays_initialized();
	} else {
		n_hit = tracer.trace(
			nerf_network,
			m_render_aabb,
			m_render_aabb_to_local,
			aabb,
			focal_length,
			nerf.cone_angle_constant,
			density_grid_bitfield,
			render_mode,
			camera_matrix1,
			depth_scale,
			m_visualized_layer,
			visualized_dimension,
			nerf.rgb_activation,
			nerf.density_activation,
			nerf.show_accel,
			nerf.max_cascade,
			nerf.render_min_transmittance,
			nerf.glow_y_cutoff,
			nerf.glow_mode,
			extra_dims_gpu,
			stream
		);
	}
	RaysNerfSoa& rays_hit = render_2d ? tracer.rays_init() : tracer.rays_hit();

	if (render_2d) {
		// Store colors in the normal buffer
		uint32_t n_elements = next_multiple(n_hit, BATCH_SIZE_GRANULARITY);
		const uint32_t floats_per_coord = sizeof(NerfCoordinate) / sizeof(float) + nerf_network->n_extra_dims();
		const uint32_t extra_stride = nerf_network->n_extra_dims() * sizeof(float); // extra stride on top of base NerfCoordinate struct

		GPUMatrix<float> positions_matrix{floats_per_coord, n_elements, stream};
		GPUMatrix<float> rgbsigma_matrix{4, n_elements, stream};

		linear_kernel(generate_nerf_network_inputs_at_current_position, 0, stream, n_hit, m_aabb, rays_hit.payload, PitchedPtr<NerfCoordinate>((NerfCoordinate*)positions_matrix.data(), 1, 0, extra_stride), extra_dims_gpu);

		if (visualized_dimension == -1) {
			nerf_network->inference(stream, positions_matrix, rgbsigma_matrix);
			linear_kernel(compute_nerf_rgba_kernel, 0, stream, n_hit, (vec4*)rgbsigma_matrix.data(), nerf.rgb_activation, nerf.density_activation, 0.01f, false);
		} else {
			nerf_network->visualize_activation(stream, m_visualized_layer, visualized_dimension, positions_matrix, rgbsigma_matrix);
		}

		linear_kernel(shade_kernel_nerf, 0, stream,
			n_hit,
			nerf.render_gbuffer_hard_edges,
			camera_matrix1,
			depth_scale,
			(vec4*)rgbsigma_matrix.data(),
			nullptr,
			rays_hit.payload,
			m_render_mode,
			nerf.training.linear_colors,
			render_buffer.frame_buffer,
			render_buffer.depth_buffer
		);
		return;
	}

	linear_kernel(shade_kernel_nerf, 0, stream,
		n_hit,
		nerf.render_gbuffer_hard_edges,
		camera_matrix1,
		depth_scale,
		rays_hit.rgba,
		rays_hit.depth,
		rays_hit.payload,
		m_render_mode,
		nerf.training.linear_colors,
		render_buffer.frame_buffer,
		render_buffer.depth_buffer
	);

	if (render_mode == ERenderMode::Cost) {
		std::vector<NerfPayload> payloads_final_cpu(n_hit);
		CUDA_CHECK_THROW(hipMemcpyAsync(payloads_final_cpu.data(), rays_hit.payload, n_hit * sizeof(NerfPayload), hipMemcpyDeviceToHost, stream));
		CUDA_CHECK_THROW(hipStreamSynchronize(stream));

		size_t total_n_steps = 0;
		for (uint32_t i = 0; i < n_hit; ++i) {
			total_n_steps += payloads_final_cpu[i].n_steps;
		}
		tlog::info() << "Total steps per hit= " << total_n_steps << "/" << n_hit << " = " << ((float)total_n_steps/(float)n_hit);
	}
}

const float* Testbed::Nerf::get_rendering_extra_dims(hipStream_t stream) const {
	CHECK_THROW(rendering_extra_dims.size() == training.dataset.n_extra_dims());

	if (training.dataset.n_extra_dims() == 0) {
		return nullptr;
	}

	const float* extra_dims_src = rendering_extra_dims_from_training_view >= 0 ?
		training.extra_dims_gpu.data() + rendering_extra_dims_from_training_view * training.dataset.n_extra_dims() :
		rendering_extra_dims.data();

	if (!training.dataset.has_light_dirs) {
		return extra_dims_src;
	}

	// the dataset has light directions, so we must construct a temporary buffer and fill it as requested.
	// we use an extra 'slot' that was pre-allocated for us at the end of the extra_dims array.
	size_t size = training.dataset.n_extra_dims() * sizeof(float);
	float* dims_gpu = training.extra_dims_gpu.data() + training.dataset.n_images * training.dataset.n_extra_dims();
	CUDA_CHECK_THROW(hipMemcpyAsync(dims_gpu, extra_dims_src, size, hipMemcpyDeviceToDevice, stream));
	vec3 light_dir = warp_direction(normalize(light_dir));
	CUDA_CHECK_THROW(hipMemcpyAsync(dims_gpu, &light_dir, min(size, sizeof(vec3)), hipMemcpyHostToDevice, stream));
	return dims_gpu;
}


std::vector<vec3> load_stl(const fs::path& path) {
	std::vector<vec3> vertices;

	std::ifstream f{native_string(path), std::ios::in | std::ios::binary};
	if (!f) {
		throw std::runtime_error{fmt::format("Mesh file '{}' not found", path.str())};
	}

	uint32_t buf[21] = {};
	f.read((char*)buf, 4 * 21);
	if (f.gcount() < 4 * 21) {
		throw std::runtime_error{fmt::format("Mesh file '{}' too small for STL header", path.str())};
	}

	uint32_t nfaces = buf[20];
	if (memcmp(buf, "solid", 5) == 0 || buf[20] == 0) {
		throw std::runtime_error{fmt::format("ASCII STL file '{}' not supported", path.str())};
	}

	vertices.reserve(nfaces * 3);
	for (uint32_t i = 0; i < nfaces; ++i) {
		f.read((char*)buf, 50);
		if (f.gcount() < 50) {
			nfaces = i;
			break;
		}

		vertices.push_back(*(vec3*)(buf + 3));
		vertices.push_back(*(vec3*)(buf + 6));
		vertices.push_back(*(vec3*)(buf + 9));
	}

	return vertices;
}

GeometryBvhNode Testbed::load_mesh(const fs::path& data_path, vec3 center) {

	GeometryBvhNode mesh_node;
    mesh_node.type = NodeType::MESH;

	tlog::info() << "Loading mesh from '" << data_path << "'";
	auto start = std::chrono::steady_clock::now();

	std::vector<vec3> vertices;
	if (equals_case_insensitive(data_path.extension(), "obj")) {
		vertices = load_obj(data_path.str());
	} else if (equals_case_insensitive(data_path.extension(), "stl")) {
		vertices = load_stl(data_path.str());
	} else {
		throw std::runtime_error{"mesh data path must be a mesh in ascii .obj or binary .stl format."};
	}

	// The expected format is
	// [v1.x][v1.y][v1.z][v2.x]...
	size_t n_vertices = vertices.size();
	size_t n_triangles = n_vertices/3;

	// Compute the AABB of the mesh
	vec3 inf(std::numeric_limits<float>::infinity());
	BoundingBox aabb (inf, inf);

	for (size_t i = 0; i < n_vertices; ++i) {
	    aabb.enlarge(vertices[i]);
	}

	// Normalize the vertices.
	for (size_t i = 0; i < n_vertices; ++i) {
	    vertices[i] = (vertices[i] - aabb.center) / max(aabb.diag());
	}

	// Store the center and scale for later use.
	// I dont think we need center! and maybe scale is not needed as well
	// if not needed delete from the struct
	mesh_node.data.mesh.mesh_center = aabb.center;
	mesh_node.data.mesh.mesh_scale = max(aabb.diag());

	// Normalize vertex coordinates to lie within [0,1]^3.
	// This way, none of the constants need to carry around
	// bounding box factors.
	for (size_t i = 0; i < n_vertices; ++i) {
		vertices[i] = (vertices[i] - aabb.min - 0.5f * aabb.diag()) / mesh_node.data.mesh.mesh_scale  + 0.5f;
	}

	mesh_node.data.mesh.triangles_cpu.resize(n_triangles);
	for (size_t i = 0; i < n_vertices; i += 3) {
		mesh_node.data.mesh.triangles_cpu[i/3] = {vertices[i+0], vertices[i+1], vertices[i+2]};
	}

	if (!mesh_node.data.mesh.triangle_bvh) {
		mesh_node.data.mesh.triangle_bvh = TriangleBvh::make();
	}

	mesh_node.data.mesh.triangle_bvh->build(mesh_node.data.mesh.triangles_cpu, 8);
	mesh_node.data.mesh.triangles_gpu.resize_and_copy_from_host(mesh_node.data.mesh.triangles_cpu);

	// initializes optix and creates OptiX program raytrace
	mesh_node.data.mesh.triangle_bvh->build_optix(mesh_node.data.mesh.triangles_gpu, m_stream.get());

	mesh_node.data.mesh.triangle_octree.reset(new TriangleOctree{});
	mesh_node.data.mesh.triangle_octree->build(*mesh_node.data.mesh.triangle_bvh, mesh_node.data.mesh.triangles_cpu, 10);

	m_bounding_radius = length(vec3(0.5f));

	// Compute discrete probability distribution for later sampling of the mesh's surface
	mesh_node.data.mesh.triangle_weights.resize(n_triangles);
	for (size_t i = 0; i < n_triangles; ++i) {
		triangle_weights.triangle_weights[i] = triangle_weights.triangles_cpu[i].surface_area();
	}
	triangle_weights.triangle_distribution.build(triangle_weights.triangle_weights);

	// Move CDF to gpu
	triangle_weights.triangle_cdf.resize_and_copy_from_host(triangle_weights.triangle_distribution.cdf);

	// Clear training data as it's no longer representative
	// of the previously loaded mesh.. but don't clear the network.
	// Perhaps it'll look interesting while morphing from one mesh to another.
	triangle_weights.training.idx = 0;
	triangle_weights.training.size = 0;

	tlog::success() << "Loaded mesh after " << tlog::durationToString(std::chrono::steady_clock::now() - start);
	tlog::info() << "  n_triangles=" << n_triangles << " aabb=" << m_raw_aabb;


	node.bb = aabb; // not sure about the correct bounding box
    node.left_idx = -1; // Set to -1 for leaf node
    node.right_idx = -1; // Set to -1 for leaf node

	return node;
}

// should I make a unique pointer and then return it?
GeometryBvhNode Testbed::load_empty_mesh_node(vec3 center) {

    GeometryBvhNode mesh_node;
    mesh_node.type = NodeType::Mesh;
	mesh_node.data.mesh = MeshData{};

	// init the meshData

    mesh_node.bb = BoundingBox{center, center+vec3(0.5f)};

    mesh_node.left_idx = -1; 
    mesh_node.right_idx = -1; 

    return mesh_node;
}


GeometryBvhNode Testbed::load_nerf(const fs::path& data_path, vec3 center) {
	
	
	GeometryBvhNode nerf_node;
    nerf_node.type = NodeType::Nerf;


	if (!data_path.empty()) {
		std::vector<fs::path> json_paths;
		if (data_path.is_directory()) {
			for (const auto& path : fs::directory{data_path}) {
				if (path.is_file() && equals_case_insensitive(path.extension(), "json")) {
					json_paths.emplace_back(path);
				}
			}
		} else if (equals_case_insensitive(data_path.extension(), "json")) {
			json_paths.emplace_back(data_path);
		} else {
			throw std::runtime_error{"NeRF data path must either be a json file or a directory containing json files."};
		}

		const auto prev_aabb_scale = nerf.training.dataset.aabb_scale;

		nerf_node.data.nerf.training.dataset = ngp::load_nerf(json_paths, nerf_node.data.nerf.sharpen);

		nerf_node.data.nerf.rgb_activation = nerf_node.data.nerf.training.dataset.is_hdr ? ENerfActivation::Exponential : ENerfActivation::Logistic;

	nerf_node.data.nerf.training.n_images_for_training = (int)nerf_node.data.nerf.training.dataset.n_images;

	nerf_node.data.nerf.training.dataset.update_metadata();

	nerf_node.data.nerf.training.cam_pos_gradient.resize(nerf_node.data.nerf.training.dataset.n_images, vec3(0.0f));
	nerf_node.data.nerf.training.cam_pos_gradient_gpu.resize_and_copy_from_host(nerf_node.data.nerf.training.cam_pos_gradient);

	nerf_node.data.nerf.training.cam_exposure.resize(nerf_node.data.nerf.training.dataset.n_images, AdamOptimizer<vec3>(1e-3f));
	nerf_node.data.nerf.training.cam_pos_offset.resize(nerf_node.data.nerf.training.dataset.n_images, AdamOptimizer<vec3>(1e-4f));
	nerf_node.data.nerf.training.cam_rot_offset.resize(nerf_node.data.nerf.training.dataset.n_images, RotationAdamOptimizer(1e-4f));
	nerf_node.data.nerf.training.cam_focal_length_offset = AdamOptimizer<vec2>(1e-5f);

	nerf_node.data.nerf.training.cam_rot_gradient.resize(nerf_node.data.nerf.training.dataset.n_images, vec3(0.0f));
	nerf_node.data.nerf.training.cam_rot_gradient_gpu.resize_and_copy_from_host(nerf_node.data.nerf.training.cam_rot_gradient);

	nerf_node.data.nerf.training.cam_exposure_gradient.resize(nerf_node.data.nerf.training.dataset.n_images, vec3(0.0f));
	nerf_node.data.nerf.training.cam_exposure_gpu.resize_and_copy_from_host(nerf_node.data.nerf.training.cam_exposure_gradient);
	nerf_node.data.nerf.training.cam_exposure_gradient_gpu.resize_and_copy_from_host(nerf_node.data.nerf.training.cam_exposure_gradient);

	nerf_node.data.nerf.training.cam_focal_length_gradient = vec2(0.0f);
	nerf_node.data.nerf.training.cam_focal_length_gradient_gpu.resize_and_copy_from_host(&nerf_node.data.nerf.training.cam_focal_length_gradient, 1);

	nerf_node.data.nerf.reset_extra_dims(m_rng);
	nerf_node.data.nerf.training.optimize_extra_dims = nerf_node.data.nerf.training.dataset.n_extra_learnable_dims > 0;

	if (nerf_node.data.nerf.training.dataset.has_rays) {
		nerf_node.data.nerf.training.near_distance = 0.0f;
	}

	nerf_node.data.nerf.training.update_transforms();

	if (!nerf_node.data.nerf.training.dataset.metadata.empty()) {
		nerf_node.data.nerf.render_lens = nerf_node.data.nerf.training.dataset.metadata[0].lens;
		m_screen_center = vec2(1.f) - nerf_node.data.nerf.training.dataset.metadata[0].principal_point;
	}

	if (!is_pot(nerf_node.data.nerf.training.dataset.aabb_scale)) {
		throw std::runtime_error{fmt::format("NeRF dataset's `aabb_scale` must be a power of two, but is {}.", nerf_node.data.nerf.training.dataset.aabb_scale)};
	}

	int max_aabb_scale = 1 << (NERF_CASCADES()-1);
	if (nerf_node.data.nerf.training.dataset.aabb_scale > max_aabb_scale) {
		throw std::runtime_error{fmt::format(
			"NeRF dataset must have `aabb_scale <= {}`, but is {}. "
			"You can increase this limit by factors of 2 by incrementing `NERF_CASCADES()` and re-compiling.",
			max_aabb_scale, nerf_node.data.nerf.training.dataset.aabb_scale
		)};
	}

	// Todo: come up with better plan!!

	BoundingBox aabb = BoundingBox{center, center};
	aabb.inflate(0.5f * std::min(1 << (NERF_CASCADES()-1), nerf_node.data.nerf.training.dataset.aabb_scale));

	m_render_aabb_to_local = nerf_node.data.nerf.training.dataset.render_aabb_to_local;	//not sure about this
	
	if (!nerf_node.data.nerf.training.dataset.render_aabb.is_empty()) {
		render_aabb = nerf_node.data.nerf.training.dataset.render_aabb.intersection(aabb);
	}

	nerf_node.data.nerf.max_cascade = 0;
	while ((1 << nerf_node.data.nerf.max_cascade) < nerf_node.data.nerf.training.dataset.aabb_scale) {
		++nerf_node.data.nerf.max_cascade;
	}

	// Perform fixed-size stepping in unit-cube scenes (like original NeRF) and exponential
	// stepping in larger scenes.
	nerf_node.data.nerf.cone_angle_constant = nerf_node.data.nerf.training.dataset.aabb_scale <= 1 ? 0.0f : (1.0f / 256.0f);

	m_up_dir = nerf_node.data.nerf.training.dataset.up;
	}

	nerf_node.bb = aabb;
    nerf_node.left_idx = -1; 
    nerf_node.right_idx = -1; 

    return nerf_node;
}

GeometryBvhNode Testbed::load_empty_nerf_node(vec3 center) {
	m_data_path = {};
	GeometryBvhNode nerf_node;
    nerf_node.type = NodeType::Nerf;
    nerf_node.data.nerf = Nerf{}; // I'm not sure if this is necessary

	nerf_node.data.nerf.training.dataset = ngp::create_empty_nerf_dataset(0, 1, false);
	load_nerf(m_data_path);
	nerf_node.data.nerf.training.n_images_for_training = 0;
    
	nerf_node.bb = BoundingBox{center, center+vec3(0.5f)};

    nerf_node.left_idx = -1; 
    nerf_node.right_idx = -1; 

    return nerf_node;
}

// future work: add a threadpool to make the loading simultaneous
void Testbed::load_scene(const fs::path& data_path) {


	/**
	 * [
    {
        "center": [0.0, 0.0, 0.0],
        "path": "path/to/geometry.obj",
        "type": "Mesh"
    },
    {
        "center": [1.0, 1.0, 1.0],
        "path": "path/to/geometry.json",
        "type": "Nerf"
    }
    // ... more geometries ...
	 *]
	 * 
	*/
	if (!data_path.empty()) {
		if (m_geometry.geometry_bvh) {
			m_geometry.geometry_bvh.reset();
		}
		if (m_geometry.bvh_nodes.size() > 0) {
			m_geometry.bvh_nodes.clear();
		}
		if (!m_geometry.geometry_bvh) {
			m_geometry.geometry_bvh = GeometryBvh::make();
		}

		std::ifstream file(data_path);
		if (!file) {
			throw std::runtime_error{fmt::format("Geometry file '{}' not found", path.str())};
		}
        
		nlohmann::json geometries = nlohmann::json::parse(file, nullptr, true, true);

        if (!geometries.is_array()) {
            throw std::runtime_error{"Geometry file must contain an array of geometry metadata."};
        }

        size_t n_nodes = geometries.size();

		m_geometry.bvh_nodes.resize(n_nodes);

		for(size_t i = 0; i < n_nodes; ++i) {
			auto& geometry = geometries[i];
            fs::path model_path = geometry["path"];

			std::string type = geometry["type"];
			std::vector<float> center = geometry["center"];
            vec3 center_vec(center[0], center[1], center[2]);

            if (type == "Mesh") {
                m_geometry.bvh_nodes[i] = load_mesh(model_path, center_vec);
            } else if (type == "Nerf") {
                m_geometry.bvh_nodes[i] = load_nerf(model_path, center_vec);
            } else {
                throw std::runtime_error{"Geometry type must be either 'Mesh' or 'Nerf'."};
            }

			// if (equals_case_insensitive(model_path.extension(), "obj") || equals_case_insensitive(model_path.extension(), "stl")) {
			// 	m_geometry.bvh_nodes[i] = load_mesh(model_path);
			// } else if (equals_case_insensitive(model_path.extension(), "json")) {
			// 	m_geometry.bvh_nodes[i] = load_nerf(model_path);
			// } else {
			// 	throw std::runtime_error{"mesh data path must be a mesh in ascii .obj or binary .stl format or nerf in json format."};
			// }
		}
	}
	
	else {

		m_geometry.bvh_nodes.resize(2);
    	
		m_geometry.bvh_nodes[0] = load_empty_mesh_node(vec3(0.0f));
    	m_geometry.bvh_nodes[1]= load_empty_nerf_node(vec3(1.0f));
		
	}
		
	
	m_geometry.geometry_bvh->build(m_geometry.geometry_bvh, 8);

	

}

}